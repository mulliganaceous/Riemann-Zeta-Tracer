#include "hip/hip_runtime.h"
// Main
#define __MAIN__

// C++ standard
#include <stdint.h>
#include <stdio.h>
#include <stdlib.h>
#include <unistd.h>
#include <iostream>
#define __STDC_FORMAT_MACROS

// CUDA runtime
#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>
#define __CUDA_RUNTIME_H__

// Complex domain
#include <complex>
#include <hip/hip_complex.h>
#define __COMPLEX

// Frame visualization
#include <opencv2/opencv.hpp>
#include <opencv2/freetype.hpp>

// Helpers
#include "helper.cu"
#include <opencv2/core/mat.hpp>

// Dimensions
#define WIDTH 256
#define HEIGHT 256
#define ENTRIES (WIDTH*HEIGHT)
#define DEPTHBLOCKBITS 2
#define DEPTHTHREADBITS 10
#define DEPTH (1 << (DEPTHBLOCKBITS + DEPTHTHREADBITS))
#define CASCADE 64
#define TERMS (DEPTH*CASCADE)
#define BATCHES 1000
#define MEMSIZE (sizeof(hipDoubleComplex) * WIDTH * HEIGHT * DEPTH)
#define OUTPUTMEMSIZE (sizeof(hipDoubleComplex) * WIDTH * HEIGHT)
#define IMGMEMSIZE (sizeof(unsigned char) * WIDTH * HEIGHT)

// CUDA Code

// Kernel definition
/*
 * Identity function
 */
__global__ void id(hipDoubleComplex *d_plot, double x_ini, double y_ini, double x_res, double y_res)
{
    // Obtain pixel subcoordinates
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int idy = blockIdx.y * blockDim.y + threadIdx.y;
    int width = gridDim.y * blockDim.y;

    // Temporary variables
    hipDoubleComplex z = make_hipDoubleComplex(x_ini + idx/x_res, y_ini + idy/y_res);
    d_plot[idx*width + idy] = z;
}

// Kernel definition
/*
 * Compute the Riemann zeta function using the Dirichlet eta function
 * without using row-reduction.
 */
__global__ void zeta(hipDoubleComplex *d_plot, hipDoubleComplex *d_input)
{
    // Obtain pixel subcoordinates
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int idy = blockIdx.y * blockDim.y + threadIdx.y;
    int width = gridDim.y * blockDim.y;

    // Temporary variables
    hipDoubleComplex z = d_input[idx*width + idy];
    hipDoubleComplex sum = make_hipDoubleComplex(0.0, 0.0);
    hipDoubleComplex temp;
    double smagnitude, angle;

    // Straightforward summation
    int terms = hipCimag(d_input[0])*1.25 + sqrt(hipCimag(d_input[0])) + log(1 + hipCimag(d_input[0]));
    if (terms < 256) {
        terms = 256;
    }
    int n;
    for (n = 1; n <= terms; n++) {
        // Must code the exponentiation manually
        smagnitude = (-1 + ((n & 1) << 1)) / exp(hipCreal(z) * log((double)n));
        angle = hipCimag(z) * log((double)n);
        temp = make_hipDoubleComplex(smagnitude * cos(angle), -smagnitude * sin(angle));
        sum = hipCadd(sum, temp);
    }

    // Average out
    smagnitude = (-1 + ((n & 1) << 1)) / exp(hipCreal(z) * log((double)n));
    angle = hipCimag(z) * log((double)n);
    temp = make_hipDoubleComplex(smagnitude * cos(angle) / 2, -smagnitude * sin(angle) / 2);
    sum = hipCadd(sum, temp);

    // Must code the coefficient manually
    temp = make_hipDoubleComplex(1 - hipCreal(z), -hipCimag(z)); // temp is now the complement of z
    smagnitude = exp(hipCreal(temp) * log(2.0));
    angle = hipCimag(temp) * log(2.0);
    temp = make_hipDoubleComplex(1 - 1 * smagnitude * cos(angle), -1 * smagnitude * sin(angle));
    sum = hipCdiv(sum, temp);

    // Store the result
    d_plot[idx*width + idy] = sum;
}

/* TODO
 * Compute the Riemann Zeta function using the Dirichlet eta function, keeping terms separate.
 */
__global__ void testterms(hipDoubleComplex *d_cube, hipDoubleComplex *d_input)
{
    // Obtain pixel subcoordinates; all threads go to the z coordinate
    int idx = blockIdx.x;
    int idy = blockIdx.y;
    int idz = blockIdx.z*blockDim.x + threadIdx.x;
    int width = gridDim.x;
    int height = gridDim.y;
    int depth = gridDim.z * blockDim.x;

    // Temporary variables
    hipDoubleComplex z = d_input[idx*height + idy];
    hipDoubleComplex sum = make_hipDoubleComplex(0.0, 0.0);
    hipDoubleComplex temp;
    double smagnitude, angle;

    // Straightforward summation
    for (int k = 1; k <= CASCADE; k++) {
        // Determine n
        int n = k + idz*CASCADE;
        sum = hipCadd(sum, z);
    }

    // Average out, only applicable for the last block and thread by depth
    if (blockIdx.z == gridDim.z - 1 && threadIdx.x == blockDim.x - 1) {
        int n = 1 + depth*CASCADE;
        sum = hipCadd(sum, z);
    }

    // Store the result, with individual terms represented as planes, and vertical lines as rows.
    d_cube[idx*height*depth + idy*depth + idz] = sum;
}

/* TODO
 * Store the Dirichlet eta terms using the Dirichlet eta function, keeping terms separate.
 */
__global__ void etaterms(hipDoubleComplex *d_cube, hipDoubleComplex *d_input)
{
    // Obtain pixel subcoordinates; all threads go to the z coordinate
    int idx = blockIdx.x;
    int idy = blockIdx.y;
    int idz = blockIdx.z*blockDim.x + threadIdx.x;
    int width = gridDim.x;
    int height = gridDim.y;
    int depth = gridDim.z * blockDim.x;

    // Temporary variables
    hipDoubleComplex z = d_input[idx*height + idy];
    hipDoubleComplex sum = make_hipDoubleComplex(0.0, 0.0);
    hipDoubleComplex temp;
    double smagnitude, angle;
    int cascade = CASCADE;// + (int)hipCimag(d_input[0])/CASCADE;
    if (cascade < CASCADE) {
        cascade = CASCADE;
    }

    // Straightforward summation
    for (int k = 1; k <= cascade; k++) {
        // Determine n
        int n = k + idz*cascade;
        // Must code the exponentiation manually
        smagnitude = (-1 + ((n & 1) << 1)) / exp(hipCreal(z) * log((double)n));
        angle = hipCimag(z) * log((double)n);
        temp = make_hipDoubleComplex(smagnitude * cos(angle), -smagnitude * sin(angle));
        sum = hipCadd(sum, temp);
    }
    __syncthreads();

    // Average out, only applicable for the last block and thread by depth
    if (blockIdx.z == gridDim.z - 1 && threadIdx.x == blockDim.x - 1) {
        int n = 1 + depth*cascade;
        smagnitude = (-1 + ((n & 1) << 1)) / exp(hipCreal(z) * log((double)n));
        angle = hipCimag(z) * log((double)n);
        temp = make_hipDoubleComplex(smagnitude * cos(angle) / 2, -smagnitude * sin(angle) / 2);
        sum = hipCadd(sum, temp);
    }
    __syncthreads();

    // Store the result, with individual terms represented as planes, and vertical lines as rows.
    d_cube[idx*height*depth + idy*depth + idz] = sum;
}

__global__ void convertEtaToZeta(hipDoubleComplex *d_plot, hipDoubleComplex *d_input) {
    // Obtain pixel subcoordinates
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int idy = blockIdx.y * blockDim.y + threadIdx.y;
    int height = gridDim.y * blockDim.y;

    hipDoubleComplex z = d_input[idx*height + idy];
    hipDoubleComplex sum = d_plot[idx*height + idy];
    hipDoubleComplex temp;
    double smagnitude, angle;

    temp = make_hipDoubleComplex(1 - hipCreal(z), -hipCimag(z)); // temp is now the complement of z
    smagnitude = exp(hipCreal(temp) * log(2.0));
    angle = hipCimag(temp) * log(2.0);
    temp = make_hipDoubleComplex(1 - smagnitude * cos(angle), -smagnitude * sin(angle));
    sum = hipCdiv(sum, temp);

    d_plot[idx*height + idy] = sum;
}

/*
 * Warp reduction for component
 */
__device__ void warpReduce(volatile double *sdata, unsigned tid) {
    sdata[tid] += sdata[tid + 32];
    sdata[tid] += sdata[tid + 16];
    sdata[tid] += sdata[tid + 8];
    sdata[tid] += sdata[tid + 4];
    sdata[tid] += sdata[tid + 2];
    sdata[tid] += sdata[tid + 1];
}

/*
 * Perform warp reduction.
 */
__global__ void warpReduceSum(hipDoubleComplex *g_idata, hipDoubleComplex *g_odata, int input_offset = 0, int output_offset = 0) {
    // Shared data is componentwise
    extern __shared__ double sdata[];

    // Load one component from global to shared. This is due to size constraints.
    unsigned idm = blockIdx.x*gridDim.y + blockIdx.y;
    unsigned tid = threadIdx.z;
    unsigned idx = (blockIdx.z*(blockDim.z << 1)) + threadIdx.z;

    // Real
    sdata[tid] = g_idata[idm*DEPTH + idx + input_offset].x + g_idata[idm*DEPTH + idx + input_offset + blockDim.z].x;
    __syncthreads();
    if (tid < 32) {
        warpReduce(sdata, tid);
    }
    if (tid == 0) {
        g_odata[idm*DEPTH + blockIdx.z + tid + output_offset].x = sdata[tid];
    }
    __syncthreads();

    // Imaginary
    sdata[tid] = g_idata[idm*DEPTH + idx + input_offset].y + g_idata[idm*DEPTH + idx + input_offset + blockDim.z].y;
    __syncthreads();
    if (tid < 32) {
        warpReduce(sdata, tid);
    }
    if (tid == 0) {
        g_odata[idm*DEPTH + blockIdx.z + tid + output_offset].y = sdata[tid];
    }
    __syncthreads();

    // Write result from shared to global
    // sdata[tid] = g_idata[idm + idx].x;
    // sdata[tid + 1] = g_idata[idm + idx].y;
    // g_odata[idm*DEPTH + blockIdx.z + tid].x = sdata[tid];
    // g_odata[idm*DEPTH + blockIdx.z + tid].y = sdata[tid];
    // if (tid == 0) {
    //     g_odata[idm*DEPTH + blockIdx.z].x = sdata[0];
    //     g_odata[idm*DEPTH + blockIdx.z].y = sdata[1];
    // }
}

/*
 * Transpose
 */
__global__ void getLayer(hipDoubleComplex *d_cube, hipDoubleComplex *d_plot, unsigned layer) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int idy = blockIdx.y * blockDim.y + threadIdx.y;
    int height = gridDim.y * blockDim.y;
    d_plot[idx*height + idy] = d_cube[idx*height*DEPTH + idy*DEPTH + layer];
}

/* Host side code */

void cudaZeta(hipDoubleComplex *h_plot, double x_ini, double y_ini, double x_res, double y_res, hipDoubleComplex *h_input)
{
    clock_t t0 = clock();
    std::cout << "Generate plot starting from height " << y_ini << std::endl;

    // Allocate device memory for the plot
    hipDoubleComplex *d_plot, *d_input;
    hipError_t status;
    status = hipHostGetDevicePointer(&d_plot, h_plot, 0);
    getStatus(status, "(d_plot) Failed to allocate hipMemcpy! ");
    status = hipHostGetDevicePointer(&d_input, h_input, 0);
    getStatus(status, "(d_plot) Failed to allocate hipMemcpy! ");
    // Perform the zeta computation
    hipDeviceSynchronize();
    id<<<dim3(WIDTH, 1), dim3(1, HEIGHT)>>>(d_input, x_ini, y_ini, x_res, y_res);
    hipDeviceSynchronize();
    zeta<<<dim3(WIDTH, 1), dim3(1, HEIGHT)>>>(d_plot, d_input);
    hipDeviceSynchronize();
    // Free memory
    hipFree(d_plot);
    hipFree(d_input);

    std::cout << "Generated plot starting from height " << y_ini << " in time " << (float)(clock() - t0)/CLOCKS_PER_SEC << "s." << std::endl;
}

/*
 * Compute the Riemann zeta function using the Dirichlet eta function, keeping terms separate.
 * Then merge the terms by depth.
 * The result is an array which applicable values are spaced DEPTH entries apart.
 */
void cudaZetaDepth(hipDoubleComplex *h_cube, hipDoubleComplex *h_sum, hipDoubleComplex *h_plot, double x_ini, double y_ini, double x_res, double y_res, hipDoubleComplex *h_input)
{
    clock_t t0;

    // Host and device-side memory allocation
    hipDoubleComplex *d_cube, *d_sum, *d_plot, *d_input;
    getStatus(hipHostGetDevicePointer(&d_cube, h_cube, 0), "(cube) Failed to allocate hipMemcpy! ");
    getStatus(hipHostGetDevicePointer(&d_sum, h_sum, 0), "(plot) Failed to allocate hipMemcpy! ");
    getStatus(hipHostGetDevicePointer(&d_plot, h_plot, 0), "(plot) Failed to allocate hipMemcpy! ");
    getStatus(hipHostGetDevicePointer(&d_input, h_input, 0), "(input) Failed to allocate hipMemcpy! ");

    // Perform the term by term computation
    t0 = clock();
    std::cout << "Generate plot starting from height " << y_ini << std::endl;
    hipDeviceSynchronize();
    id<<<dim3(WIDTH, 1), dim3(1, HEIGHT)>>>(d_input, x_ini, y_ini, x_res, y_res);
    hipDeviceSynchronize();
    etaterms<<<dim3(WIDTH, HEIGHT, 1 << DEPTHBLOCKBITS), 1 << DEPTHTHREADBITS>>>(d_cube, d_input);
    hipDeviceSynchronize();

    // Merge the terms by depth (contiguous range of 1024 blocks)
    warpReduceSum<<<dim3(WIDTH, HEIGHT, DEPTH >> 7), dim3(1, 1, 64)>>>(d_cube, d_sum, 0, 64);
    warpReduceSum<<<dim3(WIDTH, HEIGHT, DEPTH >> 7), dim3(1, 1, 64)>>>(d_sum, d_sum, 64, 0);
    hipDeviceSynchronize();
    getLayer<<<dim3(WIDTH, 1), dim3(1, HEIGHT)>>>(d_sum, d_plot, 0);
    hipDeviceSynchronize();
    convertEtaToZeta<<<dim3(WIDTH, 1), dim3(1, HEIGHT)>>>(d_plot, d_input);
    hipDeviceSynchronize();

    // Finish execution and free memory
    std::cout << "Generated plot starting from height " << y_ini << " in time " << (float)(clock() - t0)/CLOCKS_PER_SEC << "s." << std::endl;
    hipFree(d_cube);
    hipFree(d_sum);
    hipFree(d_plot);
    hipFree(d_input);
}

__global__ void generate_phase_plot(unsigned char *d_image, hipDoubleComplex *d_plot, hipDoubleComplex *d_input, int unitsquare) {
    // Obtain pixel subcoordinates
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int idy = blockIdx.y * blockDim.y + threadIdx.y;
    int width = gridDim.y * blockDim.y;

    // Input and output
    double2 z = d_input[idx*HEIGHT + idy];
    double2 zeta = d_plot[idx*HEIGHT + idy];
    double magnitude = hipCabs(zeta);
    double angle = atan2(hipCimag(zeta), hipCreal(zeta));
    double lightness = magnitude;
    double csaturation = 0.36;

    // Cross and conversion
    if (angle < 0) {
        angle += 2*M_PI;
    }
    double slope = hipCreal(zeta)/hipCimag(zeta);
    double islope = hipCimag(zeta)/hipCreal(zeta);
    unsigned char cross = ((abs(hipCreal(zeta)) < 0.0625) || (abs(slope) < 0.015625))
                        + ((abs(hipCimag(zeta)) < 0.0625) || (abs(islope) < 0.015625));

    // Output gridlines
    switch (cross) {
        case 1:
            csaturation = 0.09;
            break;
        case 2:
            csaturation = 0;
            break;
    }
    // Checkerboarding101
    if (magnitude < 32 && (((((int)(floor(zeta.x))) & 1) + (((int)(floor(zeta.y))) & 1)) & 1)) {
        csaturation = cross ? 0.25 : csaturation * 64./36.;
    }
    // Magnitude
    if (lightness > 65536) {
        lightness -= 24576; // Triangulous2
    }
    else if (lightness > 32768) {
        lightness -= 12288; // Triangulous
    }
    else if (lightness > 16384) {
        lightness -= 6144; // Volleo
    }
    else if (lightness > 8192) {
        lightness -= 3072; // Whalend
    }
    else if (lightness > 4096) {
        lightness -= 1536; // Terrence
    }
    else if (lightness > 2048) {
        lightness -= 768; // Triferatu
    }
    else if (lightness > 512) {
        lightness -= 384; // Triad
    }
    else if (lightness > 256) {
        lightness -= 192; // Threejay
    }
    else if (lightness > 128) {
        lightness -= 96; // 32 to 160
    }
    else if (lightness > 64) {
        lightness -= 48; // 16 to 80
    }
    else if (lightness > 32) {
        lightness -= 28; // 4 to 36
    }
    else if (lightness > 24) {
        lightness -= 21; // 3 to 11
    }
    else if (lightness > 16) {
        lightness -= 14; // 2 to to 10
    }
    else if (lightness > 8) {
        lightness -= 7.5 ; // 8/16 to 8.5
    }
    else if (lightness > 7) {
        lightness -= 6.5625; // 7/16
    }
    else if (lightness > 6) {
        lightness -= 5.625; // 6/16
    }
    else if (lightness > 5) {
        lightness -= 4.6875; // 5/16
    }
    else if (lightness > 4) {
        lightness -= 3.75; // 4/16
    }
    else if (lightness > 3) {
        lightness -= 2.8125; // 3/16
    }
    else if (lightness > 2) {
        lightness -= 1.875; // 2/16
    }
    else if (lightness > 1) {
        lightness -= 0.9375; // 1/16
    }

    // Output
    lightness = 0.96/(1 + 1.0/sqrtf(lightness)); // sigmoid
    // Unit circle
    if (magnitude < 1) {
        int sector = ((int)(angle*6/M_PI)) % 3;
        switch (sector) {
            case 2:
                csaturation = 0.16;
                break;
            case 1:
                csaturation = 0.09;
                break;
            case 0:
                csaturation = 0;
                break;
            default:
                csaturation = 1;
                break;
        }
    } // Antidiagonal
    else if (cross) {
        lightness /= 4;
        lightness += 0.375;
    }
    else {
        if (slope < 0) {
            csaturation = 1 - (1 - csaturation) / 2;
        }
        else {

        }
    }

    // Input
    if ((int)(hipCreal(z)) - hipCreal(z) == 0 || (int)(hipCimag(z)) - hipCimag(z) == 0) {
        lightness = lightness*lightness*lightness/sqrtf(2);
        csaturation = 1;
    }

    // Color (from NVDA)
    idx = 3*(idx*HEIGHT + idy);
    float nNormalizedH = angle/2/M_PI;
    float nNormalizedL = lightness;
    float nNormalizedS = 1 - csaturation;
    float nM1, nM2, nR, nG, nB;
    float nh = 0.0f;
    if (nNormalizedL <= 0.5F)
        nM2 = nNormalizedL * (1.0F + nNormalizedS);
    else
        nM2 = nNormalizedL + nNormalizedS - nNormalizedL * nNormalizedS;
    nM1 = 2.0F * nNormalizedL - nM2;
    if (nNormalizedS == 0.0F)
        nR = nG = nB = nNormalizedL;
    else
    {
        nh = nNormalizedH + 0.3333F;
        if (nh > 1.0F)
            nh -= 1.0F;
    }
    float nMDiff = nM2 - nM1;
    if (0.6667F < nh)
        nR = nM1;
    else
    {
        if (nh < 0.1667F)
            nR = (nM1 + nMDiff * nh * 6.0F); // / 0.1667F
        else if (nh < 0.5F)
            nR = nM2;
        else
            nR = nM1 + nMDiff * ( 0.6667F - nh ) * 6.0F; // / 0.1667F
    }
    // Green
    nh = nNormalizedH;
    if (0.6667F < nh)
        nG = nM1;
    else
    {
        if (nh < 0.1667F)
            nG = (nM1 + nMDiff * nh * 6.0F); // / 0.1667F
        else if (nh < 0.5F)
            nG = nM2;
        else
            nG = nM1 + nMDiff * (0.6667F - nh ) * 6.0F; // / 0.1667F
    }
    // Blue
    nh = nNormalizedH - 0.3333F;
    if (nh < 0.0F)
        nh += 1.0F;
    if (0.6667F < nh)
        nB = nM1;
    else
    {
        if (nh < 0.1667F)
            nB = (nM1 + nMDiff * nh * 6.0F); // / 0.1667F
        else if (nh < 0.5F)
            nB = nM2;
        else
            nB = nM1 + nMDiff * (0.6667F - nh ) * 6.0F; // / 0.1667F
    }
    d_image[idx    ] = 255*(nB);
    d_image[idx + 1] = 255*(nG);
    d_image[idx + 2] = 255*(nR);
}

void plot(hipDoubleComplex *h_plot, hipDoubleComplex *h_input, int ini, unsigned unitsquare) {
    // Generate image
    unsigned char *h_image, *d_image;
    getStatus(hipHostMalloc(&h_image, 3*IMGMEMSIZE), "(h_image) Failed to allocate hipHostMalloc! ");
    getStatus(hipHostGetDevicePointer(&d_image, h_image, 0), "(d_image) Failed to perform host to device for image");
    hipDeviceSynchronize();
    generate_phase_plot<<<dim3(WIDTH >> 5, HEIGHT >> 5), dim3(32, 32)>>>(d_image, h_plot, h_input, 32);
    hipDeviceSynchronize();
    hipFree(d_image);

    // Save image
    cv::Mat3f hls = cv::Mat(WIDTH, HEIGHT, CV_8UC3, h_image);
    std::stringstream ss;
    ss << std::setbase(10) << std::setw(4) << ini;
    std::string hexstr = ss.str();
    std::replace(hexstr.begin(), hexstr.end(), ' ', '0');
    cv::imwrite("test/plot/Plot" + hexstr + ".png", hls);

    // Draw spiral frames
    const int FINE = unitsquare / 8;
    const int YFINE = FINE / 8;
    for (int y = 0; y < HEIGHT; y += FINE/2) {
        // Generate header and spiral plot
        cv::Ptr<cv::freetype::FreeType2> ft2;
        ft2 = cv::freetype::createFreeType2();
        ft2->loadFontData("/usr/share/fonts/opentype/unifont/unifont.otf", 0 );
        double2 zeta = h_plot[WIDTH*HEIGHT/2 + y];
        double2 input = h_input[WIDTH*HEIGHT/2 + y];
        double z_y = hipCreal(input);

        // Header
        cv::Mat3b header = cv::Mat3b::zeros((1080 - HEIGHT)/2, WIDTH);
        std::stringstream headertext;
        std::stringstream decimaltext;
        decimaltext << std::setprecision(3) << std::setw(3) << z_y - (int)z_y;
        std::string hexstr = decimaltext.str();
        headertext << "zeta(0.5 + i" << (int)z_y << std::setprecision(3) << "." << hexstr.substr(2) << ") = " << (hipCreal(zeta) < 0 ? '-' : ' ') << (hipCreal(zeta) < 0 ? -hipCreal(zeta) : hipCreal(zeta)) << ' ' << (hipCimag(zeta) < 0 ? '-' : '+') << " i" << (hipCimag(zeta) < 0 ? -hipCimag(zeta) : hipCimag(zeta));
        std::string headerstr = headertext.str();
        ft2->putText(header, headerstr, cv::Point(0,16), 16, cv::Scalar(255, 255, 255), -1, cv::LINE_8, true);
        std::stringstream abstext;
        abstext << "magnitude = " << hipCabs(zeta);
        headerstr = abstext.str();
        ft2->putText(header, headerstr, cv::Point(960,16), 16, cv::Scalar(255, 255, 255), -1, cv::LINE_8, true);
        std::stringstream angletext;
        angletext << "phase = " << atan2(hipCimag(zeta), hipCreal(zeta))*180/M_PI << " deg";
        headerstr = angletext.str();
        ft2->putText(header, headerstr, cv::Point(1440,16), 16, cv::Scalar(255, 255, 255), -1, cv::LINE_8, true);
        std::stringstream ss;
        ss << std::setbase(10) << std::setw(4) << ini << ".x" << std::setbase(16) << std::setw(4) << y;
        hexstr = ss.str();
        std::replace(hexstr.begin(), hexstr.end(), ' ', '0');
        cv::imwrite("test/header/Header" + hexstr + ".gif", header);
        std::cout << " H\t" << hexstr << std::endl;

        // Spiral graph
        cv::Mat3b spiralimage = cv::Mat3b::zeros(WIDTH, HEIGHT);
        const int tracegrid = 16;
        // Vertical
        for (int t_x = WIDTH/2 - unitsquare / 2; t_x <= WIDTH/2 + unitsquare / 2; t_x += FINE) {
            for (int t_y = 0; t_y < y - 1; t_y += YFINE) {
                double2 zeta = h_plot[t_x*HEIGHT + t_y];
                double2 dzeta = h_plot[t_x*HEIGHT + t_y + YFINE];
                cv::Point2d tracezeta(HEIGHT/2 + tracegrid*hipCimag(zeta), WIDTH/2 + tracegrid*hipCreal(zeta));
                cv::Point2d tracedzeta(HEIGHT/2 + tracegrid*hipCimag(dzeta), WIDTH/2 + tracegrid*hipCreal(dzeta));
                cv::Vec3f righthalf(0, 0, 240*(1 - (((float)t_x) - WIDTH/2)/unitsquare) + 10);
                cv::Vec3f lefthalf(240*(((float)t_x) - WIDTH/2 + unitsquare)/unitsquare + 10, 0, 0);
                if (t_x < WIDTH / 2)
                    cv::line(spiralimage, tracezeta, tracedzeta, lefthalf, 1, cv::LINE_AA);
                else if (t_x > WIDTH / 2)
                    cv::line(spiralimage, tracezeta, tracedzeta, righthalf, 1, cv::LINE_AA);

            }
        }
        // Horizontal
        for (int t_x = WIDTH/2 - unitsquare / 2; t_x < WIDTH/2 + unitsquare / 2; t_x += FINE) {
            for (int t_y = 0; t_y <= y; t_y += FINE) {
                double2 zeta = h_plot[t_x*HEIGHT + t_y];
                double2 lzeta = h_plot[(t_x + FINE)*HEIGHT + t_y];
                cv::Point2d tracezeta(HEIGHT/2 + tracegrid*hipCimag(zeta), WIDTH/2 + tracegrid*hipCreal(zeta));
                cv::Point2d tracelzeta(HEIGHT/2 + tracegrid*hipCimag(lzeta), WIDTH/2 + tracegrid*hipCreal(lzeta));
                cv::Vec3f righthalf(0, 0, 240*(1 - (((float)t_x) - WIDTH/2)/unitsquare) + 10);
                cv::Vec3f lefthalf(240*(((float)t_x) - WIDTH/2 + unitsquare)/unitsquare + 10, 0, 0);
                if (t_x < WIDTH / 2)
                    cv::line(spiralimage, tracezeta, tracelzeta, lefthalf, 1, cv::LINE_AA);
                else if (t_x >= WIDTH / 2)
                    cv::line(spiralimage, tracezeta, tracelzeta, righthalf, 1, cv::LINE_AA);
            }
        }
        // Front end
        cv::Vec3f front(0,240,0);
        for (int t_x = WIDTH/2 - unitsquare/2; t_x < WIDTH/2 + unitsquare/2 + unitsquare; t_x += YFINE) {
            int t_y = y;
            double2 zeta = h_plot[t_x*HEIGHT + t_y];
            double2 lzeta = h_plot[(t_x + YFINE)*HEIGHT + t_y];
            cv::Point2d tracezeta(HEIGHT/2 + tracegrid*hipCimag(zeta), WIDTH/2 + tracegrid*hipCreal(zeta));
            cv::Point2d tracelzeta(HEIGHT/2 + tracegrid*hipCimag(lzeta), WIDTH/2 + tracegrid*hipCreal(lzeta));

            if (t_x < WIDTH / 2)
                cv::line(spiralimage, tracezeta, tracelzeta, front, 2, cv::LINE_AA);
            else if (t_x >= WIDTH / 2)
                cv::line(spiralimage, tracezeta, tracelzeta, front, 2, cv::LINE_AA);
        }
        // Basel line
        cv::Vec3f unity(0,250,0);
        int t_x = WIDTH/2 + unitsquare/2 + unitsquare;
        for (int t_y = 0; t_y < y - 1; t_y += 1) {
            double2 zeta = h_plot[t_x*HEIGHT + t_y];
            double2 dzeta = h_plot[t_x*HEIGHT + t_y + 1];
            cv::Point2d tracezeta(HEIGHT/2 + tracegrid*hipCimag(zeta), WIDTH/2 + tracegrid*hipCreal(zeta));
            cv::Point2d tracedzeta(HEIGHT/2 + tracegrid*hipCimag(dzeta), WIDTH/2 + tracegrid*hipCreal(dzeta));
            cv::line(spiralimage, tracezeta, tracedzeta, unity, 1, cv::LINE_AA);
        }
        // Critical line
        t_x = WIDTH / 2;
        cv::Vec3f criticalline(240,240,240);
        for (int t_y = 0; t_y < y - 1; t_y += 1) {
            double2 zeta = h_plot[t_x*HEIGHT + t_y];
            double2 dzeta = h_plot[t_x*HEIGHT + t_y + 1];
            cv::Point2d tracezeta(HEIGHT/2 + tracegrid*hipCimag(zeta), WIDTH/2 + tracegrid*hipCreal(zeta));
            cv::Point2d tracedzeta(HEIGHT/2 + tracegrid*hipCimag(dzeta), WIDTH/2 + tracegrid*hipCreal(dzeta));
            cv::Vec3f righthalf(480*(1-((float)t_x)/WIDTH) + 1./8, 0, 0);
            cv::Vec3f criticalline(240,240,240);
            cv::Vec3f lefthalf(0,0,240*((float)t_x)/WIDTH - 1./8);
            cv::line(spiralimage, tracezeta, tracedzeta, criticalline, 2, cv::LINE_AA);
        }
        cv::Vec3f grid(127,127,127);
        cv::Point2d realbegin(HEIGHT / 2, 0);
        cv::Point2d realend(HEIGHT/2, WIDTH);
        cv::Point2d imagbegin(0, WIDTH/2);
        cv::Point2d imagend(HEIGHT, WIDTH/2);
        cv::Point2d center(HEIGHT / 2, WIDTH / 2);
        cv::line(spiralimage, realbegin, realend, grid, 1);
        cv::line(spiralimage, imagbegin, imagend, grid, 1);
        cv::circle(spiralimage, center, tracegrid, grid, 1);

        ss = std::stringstream();
        ss << std::setbase(10) << std::setw(4) << ini << ".x" << std::setbase(16) << std::setw(4) << y;
        hexstr = ss.str();
        std::replace(hexstr.begin(), hexstr.end(), ' ', '0');

        cv :: imwrite("test/spiral/Spiral" + hexstr + ".gif", spiralimage);
        std::cout << " S\t" << hexstr << std::endl;
    }

    hipHostFree(h_image);
}

void generateplot(int initial = 0, int interval = 256, int unitsquare = 256, int increment = 4) {
    interval += initial;
    std::cout << "Generating sequences of images starting at height " << initial << ", resolution " << unitsquare << std::endl;
    for (int ini = initial; ini <= interval; ini += increment) {
        // Allocate host memory for the plot
        hipDoubleComplex *h_plot;
        getStatus(hipHostMalloc(&h_plot, OUTPUTMEMSIZE), "(h_plot) Failed to allocate hipHostMalloc! ");
        hipDoubleComplex *h_input;
        getStatus(hipHostMalloc(&h_input, OUTPUTMEMSIZE), "(h_input) Failed to allocate hipHostMalloc! ");

        // Generate plot
        double x_ini = -1.5;
        double y_ini = -1 + ini;
        cudaZeta(h_plot, x_ini, y_ini, unitsquare, unitsquare, h_input);
        plot(h_plot, h_input, ini, unitsquare);

        // Free memory
        hipHostFree(h_plot);
        hipHostFree(h_input);
    }
}

void generatedepthplot(int initial = 0, int interval = 256, int unitsquare = 256, int increment = 4) {
    interval += initial;
    std::cout << "Generating sequences of images starting at height " << initial << ", resolution " << unitsquare << std::endl;
    // Allocate host memory for the plot
    hipDoubleComplex *h_cube;
    getStatus(hipHostMalloc(&h_cube, MEMSIZE), "(h_cube) Failed to allocate hipHostMalloc! ");
    hipDoubleComplex *h_sum;
    getStatus(hipHostMalloc(&h_sum, MEMSIZE), "(h_sum) Failed to allocate hipHostMalloc! ");
    hipDoubleComplex *h_plot;
    getStatus(hipHostMalloc(&h_plot, OUTPUTMEMSIZE), "(h_plot) Failed to allocate hipHostMalloc! ");
    hipDoubleComplex *h_input;
    getStatus(hipHostMalloc(&h_input, OUTPUTMEMSIZE), "(h_input) Failed to allocate hipHostMalloc! ");
    for (int ini = initial; ini <= interval; ini += increment) {
        // Plot
        double x_ini = -1.5;
        double y_ini = -1 + ini;
        cudaZetaDepth(h_cube, h_sum, h_plot, x_ini, y_ini, unitsquare, unitsquare, h_input);

        // Test output
        if (ini == initial) {
            for (int z = 0; z < 256; z++)
            {
                std::cout << "depth " << z << std::endl;
                for (int x = 0; x < 16; x += 1)
                {
                    for (int y = 0; y < 9; y += 1)
                    {
                        std::cout << "(" << std::setw(5) << h_cube[HEIGHT * DEPTH * x + DEPTH * y + z].x << " + ";
                        std::cout <<  std::setw(5) << h_cube[HEIGHT * DEPTH * x + DEPTH * y + z].y << "j, " << ")";
                    }
                    std::cout << std::endl;
                }
            }
            for (int z = 0; z < 129; z++)
            {
                std::cout << "sum " << z << std::endl;
                for (int x = 0; x < 16; x += 1)
                {
                    for (int y = 0; y < 9; y += 1)
                    {
                        hipDoubleComplex s = h_sum[HEIGHT * DEPTH * x + DEPTH * y + z];
                        printf("(%.2f,%.2fj)", s.x, s.y);
                    }
                    std::cout << std::endl;
                }
            }
            {
                std::cout << "total " << std::endl;
                for (int x = 0; x < 16; x += 1)
                {
                    for (int y = 0; y < 9; y += 1)
                    {
                        hipDoubleComplex s = h_plot[HEIGHT * x + y];
                        printf("[%.2f,%.2fj]", s.x, s.y);
                    }
                    std::cout << std::endl;
                }
            }
        }

        // Plot
        plot(h_plot, h_input, ini, unitsquare);
    }
    // Free memory
    free(h_cube);
    free(h_sum);
    hipHostFree(h_plot);
    hipHostFree(h_input);
}

int main()
{
    // List CUDA devices
    int deviceCount;
    hipGetDeviceCount(&deviceCount);
    int deviceId;
    for (deviceId = 0; deviceId < deviceCount; deviceId++)
    {
        hipDeviceProp_t deviceProp;
        hipGetDeviceProperties(&deviceProp, deviceId);
        printf("Device %d: %s (v%d.%d)\n", deviceId, deviceProp.name, deviceProp.major, deviceProp.minor);
        printf("\tL2 cache size    : %d\n", deviceProp.l2CacheSize);
        printf("\tThread dimensions: %d,%d,%d\n", deviceProp.maxThreadsDim[0], deviceProp.maxThreadsDim[1], deviceProp.maxThreadsDim[2]);
        printf("\tMemory bus width : %d\n", deviceProp.memoryBusWidth);
        printf("\tMultiprocessor   : %d\n", deviceProp.multiProcessorCount);
        printf("\tRegisters        : %d / %d\n", deviceProp.regsPerBlock, deviceProp.regsPerMultiprocessor);
        printf("\tShared memory    : %d / %d\n", deviceProp.sharedMemPerBlock, deviceProp.sharedMemPerMultiprocessor);
        printf("\tConstant memory  : %d\n", deviceProp.totalConstMem);
        printf("\tGlobal memory    : %lld\n", deviceProp.totalGlobalMem);
        printf("\tWarp size        : %d\n", deviceProp.warpSize);
        //  printf("\tCluster support  : %d\n", deviceProp.clusterLaunch);
    }

    // Generate plot
    generatedepthplot(7000, 1024, 64, 4);

    return EXIT_SUCCESS;
}

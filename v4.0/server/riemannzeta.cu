#include "hip/hip_runtime.h"
// Main
#define __MAIN__

// C++ standard
#include <stdint.h>
#include <stdio.h>
#include <stdlib.h>
#include <unistd.h>
#include <iostream>
#define __STDC_FORMAT_MACROS

// CUDA runtime
#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>
#define __CUDA_RUNTIME_H__

// Complex domain
#include <complex>
#include <hip/hip_complex.h>
#define __COMPLEX

// Frame visualization
#include <opencv2/opencv.hpp>
#include <opencv2/freetype.hpp>

// Helpers
#include "helper.cu"
#include <opencv2/core/mat.hpp>

// Dimensions
#define WIDTH 1920
#define HEIGHT 1024
#define ENTRIES (WIDTH*HEIGHT)
#define DEPTH 16
#define CASCADE 1024
#define TERMS (DEPTH*CASCADE)
#define BATCHES 1000
#define MEMSIZE (sizeof(hipDoubleComplex) * WIDTH * HEIGHT * DEPTH)
#define IMGMEMSIZE (sizeof(unsigned char) * WIDTH * HEIGHT)

// CUDA Code

// Kernel definition
/*
 * Identity function
 */
__global__ void id(hipDoubleComplex *d_plot, double x_ini, double y_ini, double x_res, double y_res)
{
    // Obtain pixel subcoordinates
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int idy = blockIdx.y * blockDim.y + threadIdx.y;
    int width = gridDim.y * blockDim.y;

    // Temporary variables
    hipDoubleComplex z = make_hipDoubleComplex(x_ini + idx/x_res, y_ini + idy/y_res);
    d_plot[idx*width + idy] = z;
}

// Kernel definition
/*
 * Compute the Riemann zeta function using the Dirichlet eta function
 * without using row-reduction.
 */
__global__ void zeta(hipDoubleComplex *d_plot, hipDoubleComplex *d_input)
{
    // Obtain pixel subcoordinates
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int idy = blockIdx.y * blockDim.y + threadIdx.y;
    int width = gridDim.y * blockDim.y;

    // Temporary variables
    hipDoubleComplex z = d_input[idx*width + idy];
    hipDoubleComplex sum = make_hipDoubleComplex(0.0, 0.0);
    hipDoubleComplex temp;
    double smagnitude, angle;

    // Straightforward summation
    int terms = hipCimag(d_input[0])*1.25 + sqrt(hipCimag(d_input[0])) + log(1 + hipCimag(d_input[0]));
    if (terms < 256) {
        terms = 256;
    }
    int n;
    for (n = 1; n <= terms; n++) {
        // Must code the exponentiation manually
        smagnitude = (-1 + ((n & 1) << 1)) / exp(hipCreal(z) * log((double)n));
        angle = hipCimag(z) * log((double)n);
        temp = make_hipDoubleComplex(smagnitude * cos(angle), -smagnitude * sin(angle));
        sum = hipCadd(sum, temp);
    }

    // Average out
    smagnitude = (-1 + ((n & 1) << 1)) / exp(hipCreal(z) * log((double)n));
    angle = hipCimag(z) * log((double)n);
    temp = make_hipDoubleComplex(smagnitude * cos(angle) / 2, -smagnitude * sin(angle) / 2);
    sum = hipCadd(sum, temp);

    // Must code the coefficient manually
    temp = make_hipDoubleComplex(1 - hipCreal(z), -hipCimag(z)); // temp is now the complement of z
    smagnitude = exp(hipCreal(temp) * log(2.0));
    angle = hipCimag(temp) * log(2.0);
    temp = make_hipDoubleComplex(1 - 1 * smagnitude * cos(angle), -1 * smagnitude * sin(angle));
    sum = hipCdiv(sum, temp);

    // Store the result
    d_plot[idx*width + idy] = sum;
}

/* TODO
 * Compute the Riemann Zeta function using the Dirichlet eta function, keeping terms separate.
 */
__global__ void zetaterms(hipDoubleComplex *d_plot, hipDoubleComplex *d_input)
{
    // Obtain pixel subcoordinates
    int idx = blockIdx.x*blockDim.x + threadIdx.x;
    int idy = blockIdx.y*blockDim.y + threadIdx.y;
    int idz = blockIdx.z*blockDim.z + threadIdx.z;
    int width = gridDim.x * blockDim.x;
    int height = gridDim.y * blockDim.y;
    int depth = gridDim.z * blockDim.z;

    // Temporary variables
    hipDoubleComplex z = d_input[idx*width + idy];
    hipDoubleComplex sum = make_hipDoubleComplex(0.0, 0.0);
    hipDoubleComplex temp;
    double smagnitude, angle;

    // Straightforward summation
    for (int k = 1; k <= CASCADE; k++) {
        // Determine n
        int n = k + idz*CASCADE;
        // Must code the exponentiation manually
        // smagnitude = (-1 + ((n & 1) << 1)) / exp(hipCreal(z) * log((double)n));
        // angle = hipCimag(z) * log((double)n);
        // temp = make_hipDoubleComplex(smagnitude * cos(angle), -smagnitude * sin(angle));
        // sum = hipCadd(sum, temp);
        sum = make_hipDoubleComplex(n, -n);
    }

    // Average out, only applicable for the last block and thread by depth
    if (blockIdx.z == gridDim.z - 1 && threadIdx.z == blockDim.z - 1) {
        int n = 1 + depth*CASCADE;
        // smagnitude = (-1 + ((n & 1) << 1)) / exp(hipCreal(z) * log((double)n));
        // angle = hipCimag(z) * log((double)n);
        // temp = make_hipDoubleComplex(smagnitude * cos(angle) / 2, -smagnitude * sin(angle) / 2);
        // sum = hipCadd(sum, temp);
        sum = make_hipDoubleComplex(n, -n);
    }

    // Must code the coefficient manually
    // temp = make_hipDoubleComplex(1 - hipCreal(z), -hipCimag(z)); // temp is now the complement of z
    // smagnitude = exp(hipCreal(temp) * log(2.0));
    // angle = hipCimag(temp) * log(2.0);
    // temp = make_hipDoubleComplex(1 - smagnitude * cos(angle), -smagnitude * sin(angle));
    // sum = hipCdiv(sum, temp);

    // Store the result, with individual terms represented as planes, and vertical lines as rows.
    d_plot[idx*height*depth + idy*depth + idz] = sum;
}

/* TODO
 */
__device__ void warpReduce(volatile double *sdata, unsigned tid) {
    sdata[tid] = sdata[tid] + sdata[tid + 32]; 
    sdata[tid] = sdata[tid] + sdata[tid + 16];
    sdata[tid] = sdata[tid] + sdata[tid + 8];
    sdata[tid] = sdata[tid] + sdata[tid + 4];
    sdata[tid] = sdata[tid] + sdata[tid + 2];
}

/* TODO
 * Perform warp reduction.
 */
__device__ hipDoubleComplex warpReduceSum(hipDoubleComplex *g_idata, hipDoubleComplex *g_odata) {
    // Shared data is componentwise
    extern __shared__ double sdata[];

    // Load one element from global to shared. The sdata contains even-odd pairs.
    unsigned tid = threadIdx.x;
    unsigned idx = (blockIdx.x*blockDim.x << 1) + threadIdx.x;
    sdata[tid] = g_idata[idx].x + g_idata[idx + blockDim.x].x;
    sdata[tid + 1] = g_idata[idx].x + g_idata[idx + blockDim.x].x;
    __syncthreads();

    if (blockDim.x >= 512 && tid < 256){
        sdata[tid] += sdata[tid + 256];
        __syncthreads();
    }
    if (blockDim.x >= 256 && tid < 128){
        sdata[tid] += sdata[tid + 128];
        __syncthreads();
    }
    if (blockDim.x >= 128 && tid < 64){
        sdata[tid] += sdata[tid + 64];
        __syncthreads();
    }
    if (tid < 32) {
        warpReduce(sdata, tid);
    }

    // Write result from shared to global
    if (tid == 0) {
        g_odata[blockIdx.x].x = sdata[0];
        g_odata[blockIdx.x].y = sdata[0];
    }
}

void cudaZeta(hipDoubleComplex *h_plot, double x_ini, double y_ini, double x_res, double y_res, hipDoubleComplex *h_input)
{
    clock_t t0 = clock();
    std::cout << "Generate plot starting from height " << y_ini << std::endl;

    // Allocate device memory for the plot
    hipDoubleComplex *d_plot, *d_input;
    hipError_t status;
    status = hipHostGetDevicePointer(&d_plot, h_plot, 0);
    getStatus(status, "Failed to allocate hipMemcpy! ");
    status = hipHostGetDevicePointer(&d_input, h_input, 0);
    getStatus(status, "Failed to allocate hipMemcpy! ");
    // Perform the zeta computation
    hipDeviceSynchronize();
    id<<<dim3(WIDTH, 1), dim3(1, HEIGHT)>>>(d_input, x_ini, y_ini, x_res, y_res);
    hipDeviceSynchronize();
    zeta<<<dim3(WIDTH, 1), dim3(1, HEIGHT)>>>(d_plot, d_input);
    hipDeviceSynchronize();
    // Free memory
    hipFree(d_plot);
    hipFree(d_input);

    std::cout << "Generated plot starting from height " << y_ini << " in time " << (float)(clock() - t0)/CLOCKS_PER_SEC << "s." << std::endl;
}

/*
 * Compute the Riemann zeta function using the Dirichlet eta function, keeping terms separate.
 * Then merge the terms by depth.
 * The result is an array which applicable values are spaced DEPTH entries apart.
 */
void cudaZetaDepth(hipDoubleComplex *h_plot, double x_ini, double y_ini, double x_res, double y_res, hipDoubleComplex *h_input)
{
    clock_t t0;

    // Host and device-side memory allocation
    hipDoubleComplex *d_plot, *d_input;
    getStatus(hipHostGetDevicePointer(&d_plot, h_plot, 0), "(plot cube) Failed to allocate hipMemcpy! ");
    getStatus(hipHostGetDevicePointer(&d_input, h_input, 0), "(input cube) Failed to allocate hipMemcpy! ");
    // Perform the term by term computation
    t0 = clock();
    std::cout << "Generate plot starting from height " << y_ini << std::endl;
    hipDeviceSynchronize();
    id<<<dim3(WIDTH, 1), dim3(1, HEIGHT)>>>(d_input, x_ini, y_ini, x_res, y_res);
    hipDeviceSynchronize();
    zetaterms<<<dim3(WIDTH, HEIGHT, 1), DEPTH>>>(d_plot, d_input);
    hipDeviceSynchronize();
    // Merge the terms by depth (contiguous range of 1024 blocks)

    std::cout << "Generated plot starting from height " << y_ini << " in time " << (float)(clock() - t0)/CLOCKS_PER_SEC << "s." << std::endl;
    // Free memory
    hipFree(d_plot);
    hipFree(d_input);
}

__global__ void generate_phase_plot(unsigned char *d_image, hipDoubleComplex *d_plot, hipDoubleComplex *d_input, int unitsquare) {
    // Obtain pixel subcoordinates
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int idy = blockIdx.y * blockDim.y + threadIdx.y;
    int width = gridDim.y * blockDim.y;
    
    // Input and output
    double2 z = d_input[idx*HEIGHT + idy];
    double2 zeta = d_plot[idx*HEIGHT + idy];
    double magnitude = hipCabs(zeta);
    double angle = atan2(hipCimag(zeta), hipCreal(zeta));
    double lightness = magnitude;
    double csaturation = 0.36;

    // Cross and conversion
    if (angle < 0) {
        angle += 2*M_PI;
    }
    double slope = hipCreal(zeta)/hipCimag(zeta);
    double islope = hipCimag(zeta)/hipCreal(zeta);
    unsigned char cross = ((abs(hipCreal(zeta)) < 0.0625) || (abs(slope) < 0.015625))
                        + ((abs(hipCimag(zeta)) < 0.0625) || (abs(islope) < 0.015625));

    // Output gridlines
    switch (cross) {
        case 1:
            csaturation = 0.09;
            break;
        case 2:
            csaturation = 0;
            break;
    }
    // Checkerboarding
    if (magnitude < 32 && (((((int)(floor(zeta.x))) & 1) + (((int)(floor(zeta.y))) & 1)) & 1)) {
        csaturation = cross ? 0.25 : csaturation * 64./36.;
    }
    // Magnitude
    if (lightness > 65536) {
        lightness -= 24576; // Triangulous2
    }                
    else if (lightness > 32768) {
        lightness -= 12288; // Triangulous
    }                
    else if (lightness > 16384) {
        lightness -= 6144; // Volleo
    }                
    else if (lightness > 8192) {
        lightness -= 3072; // Whalend
    }                
    else if (lightness > 4096) {
        lightness -= 1536; // Terrence
    }
    else if (lightness > 2048) {
        lightness -= 768; // Triferatu
    }
    else if (lightness > 512) {
        lightness -= 384; // Triad
    }
    else if (lightness > 256) {
        lightness -= 192; // Threejay
    }
    else if (lightness > 128) {
        lightness -= 96; // 32 to 160
    }
    else if (lightness > 64) {
        lightness -= 48; // 16 to 80
    }
    else if (lightness > 32) {
        lightness -= 28; // 4 to 36
    }
    else if (lightness > 24) {
        lightness -= 21; // 3 to 11
    }
    else if (lightness > 16) {
        lightness -= 14; // 2 to to 10
    }
    else if (lightness > 8) {
        lightness -= 7.5 ; // 8/16 to 8.5
    }
    else if (lightness > 7) {
        lightness -= 6.5625; // 7/16
    }
    else if (lightness > 6) {
        lightness -= 5.625; // 6/16
    }
    else if (lightness > 5) {
        lightness -= 4.6875; // 5/16
    }
    else if (lightness > 4) {
        lightness -= 3.75; // 4/16
    }
    else if (lightness > 3) {
        lightness -= 2.8125; // 3/16
    }
    else if (lightness > 2) {
        lightness -= 1.875; // 2/16
    }
    else if (lightness > 1) {
        lightness -= 0.9375; // 1/16
    }

    // Output
    lightness = 0.96/(1 + 1.0/sqrtf(lightness)); // sigmoid
    // Unit circle
    if (magnitude < 1) {
        int sector = ((int)(angle*6/M_PI)) % 3;
        switch (sector) {
            case 2:
                csaturation = 0.16;
                break;
            case 1:
                csaturation = 0.09;
                break;
            case 0:
                csaturation = 0;
                break;
            default:
                csaturation = 1;
                break;
        }
    } // Antidiagonal
    else if (cross) {
        lightness /= 4;
        lightness += 0.375;
    }
    else {
        if (slope < 0) {
            csaturation = 1 - (1 - csaturation) / 2;
        }
        else {
            
        }
    }

    // Input
    if ((int)(hipCreal(z)) - hipCreal(z) == 0 || (int)(hipCimag(z)) - hipCimag(z) == 0) {
        lightness = lightness*lightness*lightness/sqrtf(2);
        csaturation = 1;
    }

    // Color (from NVDA)
    idx = 3*(idx*HEIGHT + idy);
    float nNormalizedH = angle/2/M_PI;
    float nNormalizedL = lightness;
    float nNormalizedS = 1 - csaturation;
    float nM1, nM2, nR, nG, nB;
    float nh = 0.0f;
    if (nNormalizedL <= 0.5F)
        nM2 = nNormalizedL * (1.0F + nNormalizedS);
    else
        nM2 = nNormalizedL + nNormalizedS - nNormalizedL * nNormalizedS;
    nM1 = 2.0F * nNormalizedL - nM2;
    if (nNormalizedS == 0.0F)
        nR = nG = nB = nNormalizedL;
    else
    {
        nh = nNormalizedH + 0.3333F;
        if (nh > 1.0F)
            nh -= 1.0F;
    }
    float nMDiff = nM2 - nM1;
    if (0.6667F < nh)
        nR = nM1;
    else
    {    
        if (nh < 0.1667F)
            nR = (nM1 + nMDiff * nh * 6.0F); // / 0.1667F
        else if (nh < 0.5F)
            nR = nM2;
        else
            nR = nM1 + nMDiff * ( 0.6667F - nh ) * 6.0F; // / 0.1667F
    }
    // Green   
    nh = nNormalizedH;
    if (0.6667F < nh)
        nG = nM1;
    else
    {
        if (nh < 0.1667F)
            nG = (nM1 + nMDiff * nh * 6.0F); // / 0.1667F
        else if (nh < 0.5F)
            nG = nM2;
        else
            nG = nM1 + nMDiff * (0.6667F - nh ) * 6.0F; // / 0.1667F
    }
    // Blue    
    nh = nNormalizedH - 0.3333F;
    if (nh < 0.0F)
        nh += 1.0F;
    if (0.6667F < nh)
        nB = nM1;
    else
    {
        if (nh < 0.1667F)
            nB = (nM1 + nMDiff * nh * 6.0F); // / 0.1667F
        else if (nh < 0.5F)
            nB = nM2;
        else
            nB = nM1 + nMDiff * (0.6667F - nh ) * 6.0F; // / 0.1667F
    }        
    d_image[idx    ] = 255*(nB);
    d_image[idx + 1] = 255*(nG);
    d_image[idx + 2] = 255*(nR);
}

void generateplot(int initial = 0, int interval = 256, int unitsquare = 256, int increment = 4) {
    interval += initial;
    std::cout << "Generating sequences of images starting at height " << initial << ", resolution " << unitsquare << std::endl;
    for (int ini = initial; ini <= interval; ini += increment) {
        // Allocate host memory for the plot
        hipDoubleComplex *h_plot;
        getStatus(hipHostMalloc(&h_plot, ENTRIES), "Failed to allocate hipHostMalloc! ");
        hipDoubleComplex *h_input;
        getStatus(hipHostMalloc(&h_input, ENTRIES), "Failed to allocate hipHostMalloc! ");

        // Plot
        double x_ini = -3.25;
        double y_ini = -2 + ini;
        cudaZeta(h_plot, x_ini, y_ini, unitsquare, unitsquare, h_input);

        // Generate image
        unsigned char *h_image, *d_image;
        getStatus(hipHostMalloc(&h_image, 3*IMGMEMSIZE), "Failed to allocate hipHostMalloc! ");
        getStatus(hipHostGetDevicePointer(&d_image, h_image, 0), "Failed to perform host to device for image");
        hipDeviceSynchronize();
        generate_phase_plot<<<dim3(WIDTH >> 5, HEIGHT >> 5), dim3(32, 32)>>>(d_image, h_plot, h_input, 32);
        hipDeviceSynchronize();
        hipFree(d_image);

        // Save image
        cv::Mat3f hls = cv::Mat(WIDTH, HEIGHT, CV_8UC3, h_image);
        std::stringstream ss;
        ss << std::setbase(10) << std::setw(4) << ini;
        std::string hexstr = ss.str();
        std::replace(hexstr.begin(), hexstr.end(), ' ', '0');
        cv::imwrite("test/plot/Plot" + hexstr + ".png", hls);

        // Draw spiral frames
        const int FINE = unitsquare / 8;
        const int YFINE = FINE / 8;
        for (int y = 0; y < HEIGHT; y += FINE/2) {
            // Generate header and spiral plot
            cv::Ptr<cv::freetype::FreeType2> ft2;
            ft2 = cv::freetype::createFreeType2();
            ft2->loadFontData("/usr/share/fonts/opentype/unifont/unifont.otf", 0 );
            double2 zeta = h_plot[WIDTH*HEIGHT/2 + y];
            double2 input = h_input[WIDTH*HEIGHT/2 + y];
            double z_y = hipCreal(input);
            
            // Header
            cv::Mat3b header = cv::Mat3b::zeros((1080 - HEIGHT)/2, WIDTH);
            std::stringstream headertext;
            std::stringstream decimaltext;
            decimaltext << std::setprecision(3) << std::setw(3) << z_y - (int)z_y;
            std::string hexstr = decimaltext.str();
            headertext << "zeta(0.5 + i" << (int)z_y << std::setprecision(3) << "." << hexstr.substr(2) << ") = " << (hipCreal(zeta) < 0 ? '-' : ' ') << (hipCreal(zeta) < 0 ? -hipCreal(zeta) : hipCreal(zeta)) << ' ' << (hipCimag(zeta) < 0 ? '-' : '+') << " i" << (hipCimag(zeta) < 0 ? -hipCimag(zeta) : hipCimag(zeta));
            std::string headerstr = headertext.str();
            ft2->putText(header, headerstr, cv::Point(0,16), 16, cv::Scalar(255, 255, 255), -1, cv::LINE_8, true);
            std::stringstream abstext;
            abstext << "magnitude = " << hipCabs(zeta);
            headerstr = abstext.str();
            ft2->putText(header, headerstr, cv::Point(960,16), 16, cv::Scalar(255, 255, 255), -1, cv::LINE_8, true);
            std::stringstream angletext;
            angletext << "phase = " << atan2(hipCimag(zeta), hipCreal(zeta))*180/M_PI << " deg";
            headerstr = angletext.str();
            ft2->putText(header, headerstr, cv::Point(1440,16), 16, cv::Scalar(255, 255, 255), -1, cv::LINE_8, true);
            std::stringstream ss;
            ss << std::setbase(10) << std::setw(4) << ini << ".x" << std::setbase(16) << std::setw(4) << y;
            hexstr = ss.str();
            std::replace(hexstr.begin(), hexstr.end(), ' ', '0');
            cv::imwrite("test/header/Header" + hexstr + ".gif", header);
            std::cout << " H\t" << hexstr << std::endl;
            
            // Spiral graph
            cv::Mat3b spiralimage = cv::Mat3b::zeros(WIDTH, HEIGHT);
            const int tracegrid = 64;
            // Vertical
            for (int t_x = WIDTH/2 - unitsquare / 2; t_x <= WIDTH/2 + unitsquare / 2; t_x += FINE) {
                for (int t_y = 0; t_y < y - 1; t_y += YFINE) {
                    double2 zeta = h_plot[t_x*HEIGHT + t_y];
                    double2 dzeta = h_plot[t_x*HEIGHT + t_y + YFINE];
                    cv::Point2d tracezeta(HEIGHT/2 + tracegrid*hipCimag(zeta), WIDTH/2 + tracegrid*hipCreal(zeta));
                    cv::Point2d tracedzeta(HEIGHT/2 + tracegrid*hipCimag(dzeta), WIDTH/2 + tracegrid*hipCreal(dzeta));
                    cv::Vec3f righthalf(0, 0, 240*(1 - (((float)t_x) - WIDTH/2)/unitsquare) + 10);
                    cv::Vec3f lefthalf(240*(((float)t_x) - WIDTH/2 + unitsquare)/unitsquare + 10, 0, 0);
                    if (t_x < WIDTH / 2)
                        cv::line(spiralimage, tracezeta, tracedzeta, lefthalf, 1, cv::LINE_AA);
                    else if (t_x > WIDTH / 2)
                        cv::line(spiralimage, tracezeta, tracedzeta, righthalf, 1, cv::LINE_AA);
                        
                }
            }
            // Horizontal
            for (int t_x = WIDTH/2 - unitsquare / 2; t_x < WIDTH/2 + unitsquare / 2; t_x += FINE) {
                for (int t_y = 0; t_y <= y; t_y += FINE) {
                    double2 zeta = h_plot[t_x*HEIGHT + t_y];
                    double2 lzeta = h_plot[(t_x + FINE)*HEIGHT + t_y];
                    cv::Point2d tracezeta(HEIGHT/2 + tracegrid*hipCimag(zeta), WIDTH/2 + tracegrid*hipCreal(zeta));
                    cv::Point2d tracelzeta(HEIGHT/2 + tracegrid*hipCimag(lzeta), WIDTH/2 + tracegrid*hipCreal(lzeta));
                    cv::Vec3f righthalf(0, 0, 240*(1 - (((float)t_x) - WIDTH/2)/unitsquare) + 10);
                    cv::Vec3f lefthalf(240*(((float)t_x) - WIDTH/2 + unitsquare)/unitsquare + 10, 0, 0);
                    if (t_x < WIDTH / 2)
                        cv::line(spiralimage, tracezeta, tracelzeta, lefthalf, 1, cv::LINE_AA);
                    else if (t_x >= WIDTH / 2)
                        cv::line(spiralimage, tracezeta, tracelzeta, righthalf, 1, cv::LINE_AA);
                }
            }
            // Front end
            cv::Vec3f front(0,240,0);
            for (int t_x = WIDTH/2 - unitsquare/2; t_x < WIDTH/2 + unitsquare/2 + unitsquare; t_x += YFINE) {
                int t_y = y;
                double2 zeta = h_plot[t_x*HEIGHT + t_y];
                double2 lzeta = h_plot[(t_x + YFINE)*HEIGHT + t_y];
                cv::Point2d tracezeta(HEIGHT/2 + tracegrid*hipCimag(zeta), WIDTH/2 + tracegrid*hipCreal(zeta));
                cv::Point2d tracelzeta(HEIGHT/2 + tracegrid*hipCimag(lzeta), WIDTH/2 + tracegrid*hipCreal(lzeta));
                
                if (t_x < WIDTH / 2)
                    cv::line(spiralimage, tracezeta, tracelzeta, front, 2, cv::LINE_AA);
                else if (t_x >= WIDTH / 2)
                    cv::line(spiralimage, tracezeta, tracelzeta, front, 2, cv::LINE_AA);
            }
            // Basel line
            cv::Vec3f unity(0,250,0);
            int t_x = WIDTH/2 + unitsquare/2 + unitsquare;
            for (int t_y = 0; t_y < y - 1; t_y += 1) {
                double2 zeta = h_plot[t_x*HEIGHT + t_y];
                double2 dzeta = h_plot[t_x*HEIGHT + t_y + 1];
                cv::Point2d tracezeta(HEIGHT/2 + tracegrid*hipCimag(zeta), WIDTH/2 + tracegrid*hipCreal(zeta));
                cv::Point2d tracedzeta(HEIGHT/2 + tracegrid*hipCimag(dzeta), WIDTH/2 + tracegrid*hipCreal(dzeta));
                cv::line(spiralimage, tracezeta, tracedzeta, unity, 1, cv::LINE_AA);
            }
            // Critical line
            t_x = WIDTH / 2;
            cv::Vec3f criticalline(240,240,240);
            for (int t_y = 0; t_y < y - 1; t_y += 1) {
                double2 zeta = h_plot[t_x*HEIGHT + t_y];
                double2 dzeta = h_plot[t_x*HEIGHT + t_y + 1];
                cv::Point2d tracezeta(HEIGHT/2 + tracegrid*hipCimag(zeta), WIDTH/2 + tracegrid*hipCreal(zeta));
                cv::Point2d tracedzeta(HEIGHT/2 + tracegrid*hipCimag(dzeta), WIDTH/2 + tracegrid*hipCreal(dzeta));
                cv::Vec3f righthalf(480*(1-((float)t_x)/WIDTH) + 1./8, 0, 0);
                cv::Vec3f criticalline(240,240,240);
                cv::Vec3f lefthalf(0,0,240*((float)t_x)/WIDTH - 1./8);
                cv::line(spiralimage, tracezeta, tracedzeta, criticalline, 2, cv::LINE_AA);
            }
            cv::Vec3f grid(127,127,127);
            cv::Point2d realbegin(HEIGHT / 2, 0);
            cv::Point2d realend(HEIGHT/2, WIDTH);
            cv::Point2d imagbegin(0, WIDTH/2);
            cv::Point2d imagend(HEIGHT, WIDTH/2);
            cv::Point2d center(HEIGHT / 2, WIDTH / 2);
            cv::line(spiralimage, realbegin, realend, grid, 1);
            cv::line(spiralimage, imagbegin, imagend, grid, 1);
            cv::circle(spiralimage, center, tracegrid, grid, 1);

            ss = std::stringstream();
            ss << std::setbase(10) << std::setw(4) << ini << ".x" << std::setbase(16) << std::setw(4) << y;
            hexstr = ss.str();
            std::replace(hexstr.begin(), hexstr.end(), ' ', '0');

            cv :: imwrite("test/spiral/Spiral" + hexstr + ".gif", spiralimage);
            std::cout << " S\t" << hexstr << std::endl;
            
        }
        // Free memory
        hipHostFree(h_plot);
        hipHostFree(h_input);
    }
}

void generatedepthplot(int initial = 0, int interval = 256, int unitsquare = 256, int increment = 4) {
    interval += initial;
    std::cout << "Generating sequences of images starting at height " << initial << ", resolution " << unitsquare << std::endl;
    for (int ini = initial; ini <= interval; ini += increment) {
        // Allocate host memory for the plot
        hipDoubleComplex *h_plot;
        getStatus(hipHostMalloc(&h_plot, MEMSIZE), "(h_plot) Failed to allocate hipHostMalloc! ");
        hipDoubleComplex *h_input;
        getStatus(hipHostMalloc(&h_input, ENTRIES), "(h_input) Failed to allocate hipHostMalloc! ");

        // Plot
        double x_ini = -3.25;
        double y_ini = -2 + ini;
        // cudaZetaDepth(h_plot, x_ini, y_ini, unitsquare, unitsquare, h_input);

        // Free memory
        hipHostFree(h_plot);
        hipHostFree(h_input);

        sleep(5);
    } 
}

int main()
{
    // List CUDA devices
    int deviceCount;
    hipGetDeviceCount(&deviceCount);
    int deviceId;
    for (deviceId = 0; deviceId < deviceCount; deviceId++)
    {
        hipDeviceProp_t deviceProp;
        hipGetDeviceProperties(&deviceProp, deviceId);
        printf("Device %d: %s (v%d.%d)\n", deviceId, deviceProp.name, deviceProp.major, deviceProp.minor);
        printf("\tL2 cache size    : %d\n", deviceProp.l2CacheSize);
        printf("\tThread dimensions: %d,%d,%d\n", deviceProp.maxThreadsDim[0], deviceProp.maxThreadsDim[1], deviceProp.maxThreadsDim[2]);
        printf("\tMemory bus width : %d\n", deviceProp.memoryBusWidth);
        printf("\tMultiprocessor   : %d\n", deviceProp.multiProcessorCount);
        printf("\tRegisters        : %d / %d\n", deviceProp.regsPerBlock, deviceProp.regsPerMultiprocessor);
        printf("\tShared memory    : %d / %d\n", deviceProp.sharedMemPerBlock, deviceProp.sharedMemPerMultiprocessor);
        printf("\tConstant memory  : %d\n", deviceProp.totalConstMem);
        printf("\tGlobal memory    : %lld\n", deviceProp.totalGlobalMem);
        printf("\tWarp size        : %d\n", deviceProp.warpSize);
        //  printf("\tCluster support  : %d\n", deviceProp.clusterLaunch);
    }

    // Generate plot
    generatedepthplot(0, 1024, 256, 16);

    return EXIT_SUCCESS;
}
